#include "hip/hip_runtime.h"
#include <thread>
#include <vector>
#include "../common/opencv-bench.h"

__constant__ uint8_t ck[100];

struct Config {
    int rows;
    int cols;
    int kernel_rows;
    int kernel_cols;
};

void erode_cuda(Mat& src, Mat& dst, Mat& kernel) {
  for (int base_row = 0; base_row < src.rows; ++base_row) {
    for (int base_col = 0; base_col < src.cols; ++base_col) {
      int len_row = std::min(kernel.rows, src.rows - base_row);
      int len_col = std::min(kernel.cols, src.cols - base_col);
      uint8_t pixel = 255;
      auto anchor = src.data + base_row * src.cols + base_col;
#pragma unroll(10)
      for (int ki = 0; ki < len_row; ++ki) {
#pragma unroll(10)
        for (int kj = 0; kj < len_col; ++kj) {
          if (kernel.data[ki * base_col + kj]) {
            pixel = std::min(pixel, anchor[ki * src.cols + kj]);
          }
        }
      }
      dst.data[base_row * src.cols + base_col] = pixel;
    }
  }
}

void erode_cuda(Mat& src, Mat& dst, Mat& kernel) {
  texture<uint8_t, cudatTextureType2D> tex8u; 
  hipCreateTextureObject()
  hipCreateTextureObject()
}


int main() {
  EXEC_CV(erode_cuda);
  return 0;
}
