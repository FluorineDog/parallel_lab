#include "hip/hip_runtime.h"

#include "../common/opencv-bench.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

struct Config {
    int rows;
    int cols;
    int kernel_rows;
    int kernel_cols;
};


//__constant__ uint8_t devKernel[10*10];
//uint8_t *devSrc;
//uint8_t *devDst;
//
//void erode_cuda(Mat& src, Mat& dst, Mat& kernel) {
//  for (int base_row = 0; base_row < src.rows; ++base_row) {
//    for (int base_col = 0; base_col < src.cols; ++base_col) {
//      int len_row = std::min(kernel.rows, src.rows - base_row);
//      int len_col = std::min(kernel.cols, src.cols - base_col);
//      uint8_t pixel = 255;
//      auto anchor = src.data + base_row * src.cols + base_col;
//#pragma unroll(10)
//      for (int ki = 0; ki < len_row; ++ki) {
//#pragma unroll(10)
//        for (int kj = 0; kj < len_col; ++kj) {
//          if (kernel.data[ki * base_col + kj]) {
//            pixel = std::min(pixel, anchor[ki * src.cols + kj]);
//          }
//        }
//      }
//      dst.data[base_row * src.cols + base_col] = pixel;
//    }
//  }
//}

void erode_cuda(Mat& src, Mat& dst, Mat& kernel) {
  int pitch;
  
//  ::hipMallocPitch(&devSrc, &pitch, src.cols, src.rows);
//  ::hipMallocPitch(&devDst, &pitch, dst.cols, dst.rows);
//  ::hipMemcpyToSymbol(HIP_SYMBOL(devKernel), kernel.data, 10*10*sizeof(uint8_t), 0);
//
//  ::hipFree(devDst);
//  ::hipFree(devSrc);
}


int main() {
  EXEC_CV(erode_cuda);
  return 0;
}
