#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define  __HIPCC__
#endif
#include "../common/opencv-bench.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_texture_types.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hipfft/hipfftXt.h>

struct Config {
    int rows;
    int cols;
    int kernel_rows;
    int kernel_cols;
};


__constant__ uint8_t kernelDev[10*10];
uint8_t *srcDev;
uint8_t *dstDev;
size_t pitch;
//texture<uint8_t, hipTextureType2D> src_tex8u;
//texture<uint8_t, hipTextureType2D> dst_tex8u;


//void erode_cuda(Mat& src, Mat& dst, Mat& kernel) {
//  for (int base_row = 0; base_row < src.rows; ++base_row) {
//    for (int base_col = 0; base_col < src.cols; ++base_col) {
//      int len_row = std::min(kernel.rows, src.rows - base_row);
//      int len_col = std::min(kernel.cols, src.cols - base_col);
//      uint8_t pixel = 255;
//      auto anchor = src.data + base_row * src.cols + base_col;
//#pragma unroll(10)
//      for (int ki = 0; ki < len_row; ++ki) {
//#pragma unroll(10)
//        for (int kj = 0; kj < len_col; ++kj) {
//          if (kernel.data[ki * base_col + kj]) {
//            pixel = std::min(pixel, anchor[ki * src.cols + kj]);
//          }
//        }
//      }
//      dst.data[base_row * src.cols + base_col] = pixel;
//    }
//  }
//}

__global__ void erode_kernel(uint8_t* src, uint8_t* dst, int pitch, int kernel_row, int kernel_col){
  __shared__ uint8_t sh_mem[64][64];
  int col = threadIdx.x;
  int row = threadIdx.y;
  int full_col = blockIdx.x * blockDim.x  + threadIdx.x;
  int full_row = blockIdx.y * blockDim.y  + threadIdx.y;
  int loc = full_row * pitch + full_col;
  bool bx = blockIdx.x == gridDim.x - 1;
  bool by = blockIdx.y == gridDim.y - 1;
  
  constexpr uint8_t border = 255;
  sh_mem[row][col] = src[loc];
  if(row < 10) sh_mem[row + 32][col] =  by ? border: src[loc + 32 * pitch];
  if(col < 10) sh_mem[row][col + 32] = bx ? border: src[loc + 32];
  if(row < 10 && col < 10) sh_mem[row + 32][col + 32] = (by || bx) ? border: src[loc + 32 + pitch * 32];
  __syncthreads();
  uint8_t pixel = border;
  for(int i = 0; i < kernel_row; ++i){
    for(int j = 0; j < kernel_col; ++j){
//      pixel = std::max(pixel, kernelDev[i * 10 + j] ? sh_mem[row + i][col + j] : border);
      uint8_t new_pixel = sh_mem[row + i][col + j];
      if( kernelDev[i * 10 + j] && new_pixel < pixel){
        pixel = new_pixel;
      }
    }
  }
  dst[loc] = pixel;
}


void erode_cuda(Mat& src, Mat& dst, Mat& kernel) {
  int cols = src.cols;
  int rows = src.rows;
  ::hipMemcpyToSymbol(HIP_SYMBOL(kernelDev), kernel.data, 10*10*sizeof(uint8_t), 0);
  ::hipMemcpy2D(srcDev, pitch,  src.data, cols, cols, rows, hipMemcpyHostToDevice);
  dim3 grids(cols / 32, rows / 32);
  dim3 blocks(32, 32);
  erode_kernel<<<grids, blocks>>>(srcDev, dstDev, pitch, kernel.rows, kernel.cols);
  
  ::hipMemcpy2D(dst.data, cols, dstDev, pitch,  cols, rows, hipMemcpyDeviceToHost);
}


int main() {
  
//  src_tex8u.addressMode[0] = src_tex8u.addressMode[1] = hipAddressModeBorder;
  
  ::hipMallocPitch(&srcDev, &pitch, 512, 512);
  
  ::hipMallocPitch(&dstDev, &pitch, 512, 512);
  
  EXEC_CV(erode_cuda);
  
  ::hipFree(dstDev);
  ::hipFree(srcDev);
  return 0;
}

