#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

__global__ void VecAdd(float* A, float* B)
{
  size_t i = threadIdx.x +  blockDim.x * blockIdx.x;
  A[i] = A[i] + B[i];
}

float *devA;
float *devB;

void cuda_add(size_t size, float A[], float B[], float C[]) {
  assert(size <= SIZE);
  ::hipMemcpy(devA, A, size, hipMemcpyHostToDevice);
  ::hipMemcpy(devB, B, size, hipMemcpyHostToDevice);
  VecAdd<<<size/1024, 1024>>>(devA, devB);
  ::hipMemcpy(C, devA, size, hipMemcpyDeviceToHost);
}

int main() {
  ::hipMalloc(&devA, SIZE * sizeof(float));
  ::hipMalloc(&devB, SIZE * sizeof(float));
  EXEC(cuda_add);
  ::hipFree(&devA);
  ::hipFree(&devB);
  return 0;
}