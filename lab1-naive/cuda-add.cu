#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

__global__ void VecAdd(float* A, float* B)
{
  size_t i = threadIdx.x +  blockDim.x * blockIdx.x;
  A[i] = A[i] + B[i];
}

float *devA;
float *devB;

void cuda_add(size_t size, float A[], float B[], float C[]) {
  assert(size <= SIZE);
  ::hipMemcpy(devA, A, size * sizeof(float), hipMemcpyHostToDevice);
  ::hipMemcpy(devB, B, size * sizeof(float), hipMemcpyHostToDevice);
  VecAdd<<< (size/256), 256 >>>(devA, devB);
  ::hipMemcpy(C, devA, size * sizeof(float), hipMemcpyDeviceToHost);
}

int main() {
  ::hipMalloc(&devA, SIZE * sizeof(float));
  ::hipMalloc(&devB, SIZE * sizeof(float));
  EXEC(cuda_add);
  ::hipFree(&devA);
  ::hipFree(&devB);
  return 0;
}