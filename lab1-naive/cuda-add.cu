#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>



void cuda_add(size_t size, float A[], float B[], float C[]) {
  for (size_t i = 0; i < size; ++i) {
    C[i] = B[i] + A[i];
  }
}

int main() {
  EXEC(cuda_add);
  return 0;
}